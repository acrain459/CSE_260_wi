#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
#include <stdio.h>

#define TW 32 // = sqrt(shared memory size / number of matrices / sizeof(_DOUBLE_)) = sqrt(0xC000 / 2 / 16)
// Runs correctly for most N (but I did not test for problems)
// Not getting improved results with below results, (N = 512, 91.6 Gflops/sec) [down from ~93 Gflops]

using namespace std;

__global__ void matMul_orig(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) 
{
	// Align Memory to 128 bits 
	/*int memsize = sizeof(_DOUBLE) * N * N;
	int alignment = 16;
	_DOUBLE_ * A_new = (_DOUBLE_ *) malloc (memsize + alignment);
	_DOUBLE_ * B_new = (_DOUBLE_ *) malloc (memsize + alignment);
	_DOUBLE_ * C_new = (_DOUBLE_ *) malloc (memsize + alignment);
	A_new += ((int) A_new % alignment) / sizeof(_DOUBLE_);
	B_new += ((int) B_new % alignment) / sizeof(_DOUBLE_);
	C_new += ((int) C_new % alignment) / sizeof(_DOUBLE_);
	memcpy (A_new, A, memsize);
	memcpy (B_new, B, memsize); //*/
	
	// Parameter Initialization
	__shared__ double As[TW][TW], Bs[TW][TW];

	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int by = blockIdx.y;
	int bx = blockIdx.x;
	int I = by * TW + ty; 
	int J =  bx * TW + tx;
	
	
    //int I =  blockIdx.y*blockDim.y + threadIdx.y;
    //int J =  blockIdx.x*blockDim.x + threadIdx.x;

    if((I < N) && (J < N))
	{
		// =================================================================
        /*_DOUBLE_ _c = 0;
		
        for (unsigned int k = 0; k < N; k++) 
		{
            _DOUBLE_ a = A_new[I * N + k];
            _DOUBLE_ b = B_new[k * N + J];
            _c += a * b;
        }
		
        C_new[I * N + J] = _c;
		// ==================================================================*/
		_DOUBLE_ Cij = 0; // should be inside of loop -_-

		// TODO: round up kk loops upper bound
		// TODO: put in conditionals to avoid array out of bounds 
		for (int kk=0; kk<N/TW; kk++) // go through each block 
		{
			// read each block into shared memory
			As[ty][tx] = A[I*N + kk*TW + tx];
			Bs[ty][tx] = B[(kk*TW + ty)*N + J];
			
			__syncthreads();
			
			// dot product 
			for (int k=0; k < TW; k++)
			{
				Cij += As[ty][k] * Bs[k][tx];
			}

			__syncthreads();
		}
		
		C[I*N + J] = Cij; // <--- this does not make sense, why update every for loop without adding?  should be outside of loop 
		// ====================================================================*/
    }
	
	//memcpy (C, C_new, memsize);
}
